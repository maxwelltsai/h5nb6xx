#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>
#include <thrust/transform.h>
#include <thrust/sequence.h>
#include <thrust/copy.h>
#include <thrust/fill.h>
#include <thrust/replace.h>
#include <thrust/functional.h>

#include "cuda_util.h"
#include "h5nb6xx_helper.h"

struct interpolate
{
    __host__ __device__
    float3 operator()(const thrust::tuple<float3, float3, float3, float3, float3, float3, float3, float3, float3>& vec)
    {
        float3 X = thrust::get<0>(vec);
        float3 V = thrust::get<1>(vec);
        float3 A = thrust::get<2>(vec);
        float3 J = thrust::get<3>(vec);
        float3 X1 = thrust::get<4>(vec);
        float3 V1 = thrust::get<5>(vec);
        float3 A1 = thrust::get<6>(vec);
        float3 J1 = thrust::get<7>(vec);
        float3 T  = thrust::get<8>(vec);

        float t = T.x;
        float t0 = T.y;
        float tstep = T.z;
        float dt = tstep;
        if (t<t0) t = t0; // fix a bug of the misalignment of HDF5 steps
        float tau = (t-t0)/tstep;
        float p0x = X.x;
        float p0y = X.y;
        float p0z = X.z;
        float p1x = V.x*dt;
        float p1y = V.y*dt;
        float p1z = V.z*dt;
        float p2x = 1.0/2*A.x*dt*dt;
        float p2y = 1.0/2*A.y*dt*dt;
        float p2z = 1.0/2*A.z*dt*dt;
        float p3x = 1.0/6*J.x*dt*dt*dt;
        float p3y = 1.0/6*J.y*dt*dt*dt;
        float p3z = 1.0/6*J.z*dt*dt*dt;
        float p4x = -1.0/6*(4*J.x+J1.x)*dt*dt*dt - 2.5*(2*A.x-A1.x)*dt*dt - 5*(4*V.x+3*V1.x)*dt - 35*(X.x-X1.x);
        float p4y = -1.0/6*(4*J.y+J1.y)*dt*dt*dt - 2.5*(2*A.y-A1.y)*dt*dt - 5*(4*V.y+3*V1.y)*dt - 35*(X.y-X1.y);
        float p4z = -1.0/6*(4*J.z+J1.z)*dt*dt*dt - 2.5*(2*A.z-A1.z)*dt*dt - 5*(4*V.z+3*V1.z)*dt - 35*(X.z-X1.z);
        float p5x = 0.5*(2*J.x+J1.x)*dt*dt*dt + (10*A.x-7*A1.x)*dt*dt + 3*(15*V.x+13*V1.x)*dt + 84*(X.x-X1.x);
        float p5y = 0.5*(2*J.y+J1.y)*dt*dt*dt + (10*A.y-7*A1.y)*dt*dt + 3*(15*V.y+13*V1.y)*dt + 84*(X.y-X1.y);
        float p5z = 0.5*(2*J.z+J1.z)*dt*dt*dt + (10*A.z-7*A1.z)*dt*dt + 3*(15*V.z+13*V1.z)*dt + 84*(X.z-X1.z);
        float p6x = -1.0/6*(4*J.x+3*J1.x)*dt*dt*dt - 0.5*(15*A.x-13*A1.x)*dt*dt - 2*(18*V.x+17*V1.x)*dt - 70*(X.x-X1.x);
        float p6y = -1.0/6*(4*J.y+3*J1.y)*dt*dt*dt - 0.5*(15*A.y-13*A1.y)*dt*dt - 2*(18*V.y+17*V1.y)*dt - 70*(X.y-X1.y);
        float p6z = -1.0/6*(4*J.z+3*J1.z)*dt*dt*dt - 0.5*(15*A.z-13*A1.z)*dt*dt - 2*(18*V.z+17*V1.z)*dt - 70*(X.z-X1.z);
        float p7x = 1.0/6*(J.x+J1.x)*dt*dt*dt + 2*(A.x-A1.x)*dt*dt + 10*(V.x+V1.x)*dt + 20*(X.x-X1.x);
        float p7y = 1.0/6*(J.y+J1.y)*dt*dt*dt + 2*(A.y-A1.y)*dt*dt + 10*(V.y+V1.y)*dt + 20*(X.y-X1.y);
        float p7z = 1.0/6*(J.z+J1.z)*dt*dt*dt + 2*(A.z-A1.z)*dt*dt + 10*(V.z+V1.z)*dt + 20*(X.z-X1.z);

        float x_pred = p0x + p1x*tau + p2x*pow(tau,2.0f) + p3x*pow(tau,3.0f) + p4x*pow(tau,4.0f) + p5x*pow(tau,5.0f) + p6x*pow(tau,6.0f) + p7x*pow(tau,7.0f);
        float y_pred = p0y + p1y*tau + p2y*pow(tau,2.0f) + p3y*pow(tau,3.0f) + p4y*pow(tau,4.0f) + p5y*pow(tau,5.0f) + p6y*pow(tau,6.0f) + p7y*pow(tau,7.0f);
        float z_pred = p0z + p1z*tau + p2z*pow(tau,2.0f) + p3z*pow(tau,3.0f) + p4z*pow(tau,4.0f) + p5z*pow(tau,5.0f) + p6z*pow(tau,6.0f) + p7z*pow(tau,7.0f);

        // fail-safe linear interpolation based on positions in case the v, a, j vectors are wrong
        // assuming that the interpolated x should be x0 <= x_interp <= x1
        //if ((x_pred<X.x||x_pred>X1.x) || (y_pred<X.y||y_pred>X1.y) || (z_pred<X.z||z_pred>X1.z)) {
        //    printf("crazy!!, x0=%f, x1=%f, x_pred=%f, t0=%f, t=%f, tau=%f\n", X.x, X1.x, x_pred, t0, t, tau);
        //    x_pred = X.x + tau * (X1.x - X.x);
        //    y_pred = X.y + tau * (X1.y - X.y);
        //    z_pred = X.z + tau * (X1.z - X.z);
        //}
        return make_float3(x_pred, y_pred, z_pred);
    }
};

struct tuple_to_float3 {
    __host__ __device__
    float3 operator()(thrust::tuple<float, float, float> vec) {
        float x = thrust::get<0>(vec);
        float y = thrust::get<1>(vec);
        float z = thrust::get<2>(vec);
        return make_float3(x, y, z);
    }
};

CUDA_Util::CUDA_Util(H5nb6xx_Helper* helper){
    this->h5nb6xx_helper = helper;
}
/*
int CUDA_Util::cuda_load_data() {

    return 0;
}

int CUDA_Util::cuda_free_data() {
    return 0;
}
*/

int CUDA_Util::cuda_predict(float to_time){
    H5nb6xx_Helper::Status istatus = this->h5nb6xx_helper->get_status();
    H5nb6xx_Helper::Dynamics* idata = this->h5nb6xx_helper->get_data();
    H5nb6xx_Helper::Dynamics* idata1 = this->h5nb6xx_helper->get_data_next();

    int n_particles = istatus.n_particles;
    float current_time = idata->time;
    std::cout<<"interpolation, dt="<<(to_time - current_time)<<" t0="<<current_time<<" t="<<to_time<<" t1="<<idata1->time<<std::endl;
    thrust::device_vector<float> x(idata->x, idata->x + n_particles);
    thrust::device_vector<float> y(idata->y, idata->y + n_particles);
    thrust::device_vector<float> z(idata->z, idata->z + n_particles);
    thrust::device_vector<float> vx(idata->vx, idata->vx + n_particles);
    thrust::device_vector<float> vy(idata->vy, idata->vy + n_particles);
    thrust::device_vector<float> vz(idata->vz, idata->vz + n_particles);
    thrust::device_vector<float> ax(idata->ax, idata->ax + n_particles);
    thrust::device_vector<float> ay(idata->ay, idata->ay + n_particles);
    thrust::device_vector<float> az(idata->az, idata->az + n_particles);
    thrust::device_vector<float> jx(idata->jx, idata->jx + n_particles);
    thrust::device_vector<float> jy(idata->jy, idata->jy + n_particles);
    thrust::device_vector<float> jz(idata->jz, idata->jz + n_particles);

    thrust::device_vector<float> x1(idata1->x, idata1->x + n_particles);
    thrust::device_vector<float> y1(idata1->y, idata1->y + n_particles);
    thrust::device_vector<float> z1(idata1->z, idata1->z + n_particles);
    thrust::device_vector<float> vx1(idata1->vx, idata1->vx + n_particles);
    thrust::device_vector<float> vy1(idata1->vy, idata1->vy + n_particles);
    thrust::device_vector<float> vz1(idata1->vz, idata1->vz + n_particles);
    thrust::device_vector<float> ax1(idata1->ax, idata1->ax + n_particles);
    thrust::device_vector<float> ay1(idata1->ay, idata1->ay + n_particles);
    thrust::device_vector<float> az1(idata1->az, idata1->az + n_particles);
    thrust::device_vector<float> jx1(idata1->jx, idata1->jx + n_particles);
    thrust::device_vector<float> jy1(idata1->jy, idata1->jy + n_particles);
    thrust::device_vector<float> jz1(idata1->jz, idata1->jz + n_particles);

    thrust::device_vector<float3> X(n_particles);
    thrust::device_vector<float3> V(n_particles);
    thrust::device_vector<float3> A(n_particles);
    thrust::device_vector<float3> J(n_particles);
    thrust::device_vector<float3> X1(n_particles);
    thrust::device_vector<float3> V1(n_particles);
    thrust::device_vector<float3> A1(n_particles);
    thrust::device_vector<float3> J1(n_particles);
    thrust::device_vector<float3> T(n_particles);
    thrust::device_vector<float> t(n_particles);
    thrust::device_vector<float> t0(n_particles);
    thrust::device_vector<float> tstep(n_particles);

    thrust::host_vector<float3> X_h(n_particles);

    thrust::fill(t.begin(), t.end(), to_time);
    thrust::fill(t0.begin(), t0.end(), current_time);
    thrust::fill(tstep.begin(), tstep.end(), istatus.t_step);

    std::cout<<"t="<<to_time<<" t0="<<current_time<<" tstep="<<istatus.t_step<<" tau="<<((to_time-current_time)/istatus.t_step)<<std::endl;

    thrust::transform(thrust::make_zip_iterator(thrust::make_tuple(x.begin(), y.begin(), z.begin())),
            thrust::make_zip_iterator(thrust::make_tuple(x.end(), y.end(), z.end())),
            X.begin(), 
            tuple_to_float3());

    thrust::transform(thrust::make_zip_iterator(thrust::make_tuple(vx.begin(), vy.begin(), vz.begin())),
            thrust::make_zip_iterator(thrust::make_tuple(vx.end(), vy.end(), vz.end())),
            V.begin(), 
            tuple_to_float3());

    thrust::transform(thrust::make_zip_iterator(thrust::make_tuple(ax.begin(), ay.begin(), az.begin())),
            thrust::make_zip_iterator(thrust::make_tuple(ax.end(), ay.end(), az.end())),
            A.begin(), 
            tuple_to_float3());

    thrust::transform(thrust::make_zip_iterator(thrust::make_tuple(jx.begin(), jy.begin(), jz.begin())),
            thrust::make_zip_iterator(thrust::make_tuple(jx.end(), jy.end(), jz.end())),
            J.begin(), 
            tuple_to_float3());

    thrust::transform(thrust::make_zip_iterator(thrust::make_tuple(t.begin(), t0.begin(), tstep.begin())),
            thrust::make_zip_iterator(thrust::make_tuple(t.end(), t0.end(), tstep.end())),
            T.begin(), 
            tuple_to_float3());

    thrust::transform(thrust::make_zip_iterator(thrust::make_tuple(x1.begin(), y1.begin(), z1.begin())),
            thrust::make_zip_iterator(thrust::make_tuple(x1.end(), y1.end(), z1.end())),
            X1.begin(), 
            tuple_to_float3());

    thrust::transform(thrust::make_zip_iterator(thrust::make_tuple(vx1.begin(), vy1.begin(), vz1.begin())),
            thrust::make_zip_iterator(thrust::make_tuple(vx1.end(), vy1.end(), vz1.end())),
            V1.begin(), 
            tuple_to_float3());

    thrust::transform(thrust::make_zip_iterator(thrust::make_tuple(ax1.begin(), ay1.begin(), az1.begin())),
            thrust::make_zip_iterator(thrust::make_tuple(ax1.end(), ay1.end(), az1.end())),
            A1.begin(), 
            tuple_to_float3());

    thrust::transform(thrust::make_zip_iterator(thrust::make_tuple(jx1.begin(), jy1.begin(), jz1.begin())),
            thrust::make_zip_iterator(thrust::make_tuple(jx1.end(), jy1.end(), jz1.end())),
            J1.begin(), 
            tuple_to_float3());

    thrust::transform(thrust::make_zip_iterator(thrust::make_tuple(t.begin(), t0.begin(), tstep.begin())),
            thrust::make_zip_iterator(thrust::make_tuple(t.end(), t0.end(), tstep.end())),
            T.begin(), 
            tuple_to_float3());

    thrust::transform(thrust::make_zip_iterator(thrust::make_tuple(X.begin(), V.begin(), A.begin(), J.begin(), X1.begin(), V1.begin(), A1.begin(), J1.begin(), T.begin())),
            thrust::make_zip_iterator(thrust::make_tuple(X.end(), V.end(), A.end(), J.end(), X1.end(), V1.end(), A1.end(), J1.end(), T.end())),
            X.begin(),
            interpolate());

    thrust::copy(X.begin(), X.end(), X_h.begin());


    for(int i=0; i<n_particles;i++) {
        idata->x[i] = X_h[i].x;
        idata->y[i] = X_h[i].y;
        idata->z[i] = X_h[i].z;
    }

    // clean up memory
    x.clear();
    y.clear();
    z.clear();
    vx.clear();
    vy.clear();
    vz.clear();
    ax.clear();
    ay.clear();
    az.clear();
    jx.clear();
    jy.clear();
    jz.clear();
    x1.clear();
    y1.clear();
    z1.clear();
    vx1.clear();
    vy1.clear();
    vz1.clear();
    ax1.clear();
    ay1.clear();
    az1.clear();
    jx1.clear();
    jy1.clear();
    jz1.clear();
    X.clear();
    V.clear();
    A.clear();
    J.clear();
    X1.clear();
    V1.clear();
    A1.clear();
    J1.clear();
    T.clear();
    t.clear();
    t0.clear();
    tstep.clear();
    X_h.clear();

    //delete istatus;
    //delete idata;
    //delete idata1;
    return 0;
}


int CUDA_Util::cuda_get_acceleration(int* x, int* y, int* z, int n_points, float time) {

    return 0;
}

int CUDA_Util::cuda_get_potential(int* x, int* y, int* z, int n_points, float time) {

    return 0;
}

